#include "hip/hip_runtime.h"
#include "../chubarov.h"

__global__ void EvalGradLMulKernel(std::size_t N,
                                   std::size_t M,
                                   std::size_t L, 
                                   float* grads,
                                   const float* sibling_values,
                                   const float* parent_grads) {

    std::size_t n = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t l = blockIdx.y * blockDim.y + threadIdx.y;

    if (n < N && l < L) {
        float grads_value = 0.0f;

        for (std::size_t m = 0; m < M; m++) {
            float parent_grad =  parent_grads[n * M + m];
            float local_grad = sibling_values[l * M + m];
            grads_value += local_grad * parent_grad;
        }
        grads[n * L + l] += grads_value;
    }
}

// FIXME: errorcheck!!
int Chubarov_EvalGradLMul(std::size_t N,
                          std::size_t M,
                          std::size_t L, 
                          float* grads,
                          const float* sibling_values,
                          const float* parent_grads) {

    float* d_sibling_values = nullptr;
    float* d_parent_grads   = nullptr;
    float* d_grads          = nullptr;

    std::size_t sibling_size = L * M * sizeof(float);
    std::size_t parent_size  = N * M * sizeof(float);
    std::size_t grads_size   = N * L * sizeof(float);

    hipMalloc(&d_sibling_values, sibling_size);
    hipMalloc(&d_parent_grads,    parent_size);
    hipMalloc(&d_grads,            grads_size);

    hipMemcpy(d_sibling_values, sibling_values, sibling_size, hipMemcpyHostToDevice);
    hipMemcpy(d_parent_grads,   parent_grads,    parent_size, hipMemcpyHostToDevice);
    hipMemcpy(d_grads,          grads,            grads_size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (L + blockDim.y - 1) / blockDim.y);

    EvalGradLMulKernel<<<gridDim, blockDim>>>(N, M, L, d_grads, d_sibling_values, d_parent_grads);

    hipMemcpy(grads, d_grads, grads_size, hipMemcpyDeviceToHost);

    hipFree(d_sibling_values);
    hipFree(d_parent_grads);
    hipFree(d_grads);

    return 0;
}


__global__ void EvalGradRMulKernel(float* sibling_values, float* parent_grads, float* grads, std::size_t N, std::size_t M, std::size_t L) {
    std::size_t l = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t m = blockIdx.y * blockDim.y + threadIdx.y;

    if (l < L && m < M) {
        float grad_value = 0.0f;
        for (std::size_t n = 0; n < N; n++) {
            float local_grad = sibling_values[n * L + l];
            float parent_grad = parent_grads[n * M + m];
            grad_value += local_grad * parent_grad;
        }
        grads[l * M + m] = grad_value;
    }
}

int Chubarov_EvalGradRMul(std::size_t N,
                           std::size_t M,
                           std::size_t L, 
                           float* grads,
                           const float* sibling_values,
                           const float* parent_grads) {
    float* d_sibling_values = nullptr;
    float* d_parent_grads   = nullptr;
    float* d_grads          = nullptr;

    std::size_t sibling_size = N * L * sizeof(float);
    std::size_t parent_size  = N * M * sizeof(float);
    std::size_t grads_size   = L * M * sizeof(float);

    hipMalloc(&d_sibling_values, sibling_size);
    hipMalloc(&d_parent_grads,    parent_size);
    hipMalloc(&d_grads,            grads_size);

    hipMemcpy(d_sibling_values, sibling_values, sibling_size, hipMemcpyHostToDevice);
    hipMemcpy(d_parent_grads,     parent_grads,  parent_size, hipMemcpyHostToDevice);
    hipMemcpy(d_grads,                   grads,   grads_size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16); // Adjust according to your needs
    dim3 gridDim((L + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    EvalGradRMulKernel<<<gridDim, blockDim>>>(d_sibling_values, d_parent_grads, d_grads, N, M, L);

    hipMemcpy(grads, d_grads, grads_size, hipMemcpyDeviceToHost);

    hipFree(d_sibling_values);
    hipFree(d_parent_grads);
    hipFree(d_grads);

    return 0;
}


__global__ void MulKernel(std::size_t N, 
                          std::size_t M,
                          std::size_t L, 
                          float* output,
                          const float* first,
                          const float* second) {
    std::size_t n = blockIdx.x * blockDim.x + threadIdx.x;
    std::size_t m = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (n < N && m < M) {
        float value = 0.0f;
        for (std::size_t l = 0; l < L; l++) {
            value += first[n * L + l] * second[l * M + m];
        }
        output[n * M + m] = value;
    }
}


int Chubarov_Mul(std::size_t N,
                 std::size_t M,
                 std::size_t L, 
                 float* output,
                 const float* first,
                 const float* second) {
    float* d_first  = nullptr;
    float* d_second = nullptr;
    float* d_output = nullptr;

    std::size_t first_size  = N * L * sizeof(float);
    std::size_t second_size = L * M * sizeof(float);
    std::size_t output_size = N * M * sizeof(float);

    hipMalloc(&d_first,   first_size);
    hipMalloc(&d_second, second_size);
    hipMalloc(&d_output, output_size);

    hipMemcpy(d_first,   first,  first_size, hipMemcpyHostToDevice);
    hipMemcpy(d_second, second, second_size, hipMemcpyHostToDevice);
    hipMemcpy(d_output, output, output_size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    MulKernel<<<gridDim, blockDim>>>(N, M, L, d_output, d_first, d_second);

    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);

    hipFree(d_first);
    hipFree(d_second);
    hipFree(d_output);

    return 0;
}
